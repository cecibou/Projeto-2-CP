#include "hip/hip_runtime.h"
// Block.cu
#include "Block.h"
#include "sha256.h"
#include <iostream>

__global__ void mineBlockKernel(uint32_t difficulty, uint32_t* nonce, char* hash)
{
    char cstr[difficulty + 1];
    for (uint32_t i = 0; i < difficulty; ++i)
    {
        cstr[i] = '0';
    }
    cstr[difficulty] = '\0';

    string str(cstr);

    do
    {
        atomicAdd(nonce, 1);
        stringstream ss;
        ss << blockIdx.x << sPrevHash << time(nullptr) << _sData << *nonce;
        sha256(ss.str().c_str(), hash);

    } while (hash[0] != str[0] || hash[1] != str[1] || hash[2] != str[2]); // Adapt this for your difficulty

    std::cout << "Block mined: " << hash << std::endl;
}

Block::Block(uint32_t nIndexIn, const string& sDataIn) : _nIndex(nIndexIn), _sData(sDataIn)
{
    _nNonce = 0;
    _tTime = time(nullptr);
    sHash = _CalculateHash();
}

void Block::MineBlock(uint32_t nDifficulty)
{
    char* d_hash;
    uint32_t* d_nonce;

    hipMalloc((void**)&d_hash, SHA256_BLOCK_SIZE);
    hipMalloc((void**)&d_nonce, sizeof(uint32_t));

    hipMemcpy(d_hash, sHash.c_str(), SHA256_BLOCK_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(d_nonce, &_nNonce, sizeof(uint32_t), hipMemcpyHostToDevice);

    mineBlockKernel<<<1, 1>>>(nDifficulty, d_nonce, d_hash);

    hipMemcpy(sHash.data(), d_hash, SHA256_BLOCK_SIZE, hipMemcpyDeviceToHost);
    hipMemcpy(&_nNonce, d_nonce, sizeof(uint32_t), hipMemcpyDeviceToHost);

    hipFree(d_hash);
    hipFree(d_nonce);
}

inline string Block::_CalculateHash() const
{
    stringstream ss;
    ss << _nIndex << sPrevHash << _tTime << _sData << _nNonce;
    return sha256(ss.str());
}
